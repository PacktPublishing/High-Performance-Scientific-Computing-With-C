#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


// Calculate an estimated value of pi using n random Monte-Carlo draws
__global__ void estimate_pi(int seed, int per_thread, hiprandState *state, unsigned int *result) 
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init((seed << 20) + id, 0, 0, &state[id]);
    float r,x,y;
    int i;
    for(i=0;i<per_thread;i++)
    {
        x = hiprand_uniform(&state[id]);
        y = hiprand_uniform(&state[id]);
        r = sqrtf(x*x+y*y);
        result[id] += (r < 1) ? 1 : 0;
    }
}

// Sum all the results together.
void serial_reduce(int n, unsigned int *in, unsigned long *out)
{
    int i;
    *out = 0;
    for(i=0;i<n;i++)
    {
        *out += in[i];
    }
}


int main(int argc, char * argv[])
{
    if (argc < 2)
    {
        printf("Please input number of sample points \n");
        exit(-1);
    }
    float n;
    int threads=512; 
    int blocks=2048;
    sscanf(argv[1], "%e", &n);
    n=n-(n%threads*blocks); //get a nice number of points to run on our GPU
    if (n < threads*blocks) n=threads*blocks; // run at least enough points to use the GPU
    int per_thread = n/(threads*blocks);
    unsigned int *pi; 
    printf("%d %d %d\n", blocks, threads, per_thread);
    hipMallocManaged(&pi, threads*blocks*sizeof(int));
    
    hiprandState_t* state;
    hipMalloc(&state, threads*blocks*sizeof(hiprandState_t));
    estimate_pi<<<blocks, threads>>>(time(NULL), per_thread, state, pi);

    unsigned long *sum_pi; 
    hipMallocManaged(&sum_pi, sizeof(long));

    hipDeviceSynchronize();
    serial_reduce(threads*blocks, pi, sum_pi);
    printf("Pi estimate: %7.6f\n", (float) 4.*(*sum_pi)/n);
    hipFree(state);
    return 0;
}
